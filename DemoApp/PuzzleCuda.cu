#include "hip/hip_runtime.h"
#include "PuzzleCuda.cuh"

__global__ void ComputeSpatialDistancePixelSeeds(SeedSuperPixel* seeds, int* closestSeed, int width, int height, int Nseeds)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int offset = x + y * width;

	float minDist = 100000.0f;
	float distSpatial = 0;
	int minSeedIndex = 0;
	if (x >= 0 && x < width && y >= 0 && y < height){
		for (int k = 0; k < Nseeds; k++){
			distSpatial = sqrtf((x - seeds[k].posX) * (x - seeds[k].posX) + (y - seeds[k].posY) * (y - seeds[k].posY));
			if (distSpatial < minDist){
				minDist = distSpatial;
				minSeedIndex = k;
			}
		}
		closestSeed[offset] = minSeedIndex;
	}
}

__global__ void ComputeDistancePixelSeeds(unsigned char* rgbData, SeedSuperPixel* seeds, int* closestSeed, int width, int height, int Nseeds, float ratio)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int offset = x + y * width;

	float minDist = 100000.0f;
	float distSpatial = 0;
	float distColor = 0;
	float dist = 0;
	int minSeedIndex = 0;

	if (x >= 0 && x < width && y >= 0 && y < height)
	{
		int m = 50;
		for (int k = 0; k < Nseeds; k++)
		{
			distSpatial = sqrtf((x - seeds[k].posX) * (x - seeds[k].posX) + (y - seeds[k].posY) * (y - seeds[k].posY));
			if (distSpatial < seeds[k].dimSuperPixel)
			{
				distColor = sqrtf(
					(rgbData[3 * offset] - seeds[k].color[0]) * (rgbData[3 * offset] - seeds[k].color[0]) +
					(rgbData[3 * offset + 1] - seeds[k].color[1]) * (rgbData[3 * offset + 1] - seeds[k].color[1]) +
					(rgbData[3 * offset + 2] - seeds[k].color[2]) * (rgbData[3 * offset + 2] - seeds[k].color[2]));
				dist = distColor + ratio * distSpatial;
				if (dist < minDist)
				{
					minDist = dist;
					minSeedIndex = k;
				}
			}

		}
		closestSeed[offset] = minSeedIndex;
	}
}

__global__ void UpdateSeeds(unsigned char* rgbData, int* closestSeed, SeedSuperPixel* newSeeds, int width, int height, int Nseed)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int offset = x + y * width;
	int index = 0;
	if (x >= 0 && x < width && y >= 0 && y < height)
	{
		index = closestSeed[offset];
		atomicAdd(&(newSeeds[index].posX), x);
		atomicAdd(&(newSeeds[index].posY), y);
		atomicAdd(&(newSeeds[index].nbPixel), 1);
		atomicAdd(&(newSeeds[index].color[0]), rgbData[3 * offset]);
		atomicAdd(&(newSeeds[index].color[1]), rgbData[3 * offset + 1]);
		atomicAdd(&(newSeeds[index].color[2]), rgbData[3 * offset + 2]);
	}
}

__global__ void ResetSeeds(SeedSuperPixel* seeds, int N)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	if (x >= 0 && x < N)
	{
		seeds[x].posX = 0;
		seeds[x].posY = 0;
		seeds[x].nbPixel = 0;
		seeds[x].color[0] = 0;
		seeds[x].color[1] = 0;
		seeds[x].color[2] = 0;
	}
}

__global__ void AverageSeeds(SeedSuperPixel* seeds, int N)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	if (x >= 0 && x < N)
	{
		seeds[x].posX /= seeds[x].nbPixel;
		seeds[x].posY /= seeds[x].nbPixel;
		seeds[x].color[0] /= seeds[x].nbPixel;
		seeds[x].color[1] /= seeds[x].nbPixel;
		seeds[x].color[2] /= seeds[x].nbPixel;
	}
}

void ComputeDistancePixelFromSeeds(cv::Mat puzzleImage, SeedSuperPixel* seeds, int* pixelSuperSeed, const int width, const int height, const int Nseeds, const float ratio, const int nbLoop)
{
	unsigned char* d_puzzleImageRGB;
	hipMalloc((void**)&d_puzzleImageRGB, (size_t)puzzleImage.cols * (size_t)puzzleImage.rows * 3);
	hipMemcpy(d_puzzleImageRGB, puzzleImage.data, (size_t)puzzleImage.cols * (size_t)puzzleImage.rows * 3, hipMemcpyHostToDevice);

	SeedSuperPixel* d_seeds;
	hipMalloc((void**)&d_seeds, sizeof(SeedSuperPixel) * Nseeds);
	hipMemcpy(d_seeds, seeds, sizeof(SeedSuperPixel) * Nseeds, hipMemcpyHostToDevice);

	int* d_closestSeed;
	hipMalloc((void**)&d_closestSeed, width * height * sizeof(int));
	hipMemset(d_closestSeed, 0, width * height * sizeof(int));

	dim3 threads(16, 16);
	dim3 blocks( (width + threads.x - 1)/ (threads.x), (height + threads.y - 1)/ (threads.y));

	dim3 threadsReset(16);
	dim3 blocksReset((Nseeds + threadsReset.x - 1) / threadsReset.x);

	for (int i = 0; i < nbLoop; i++){
		ComputeDistancePixelSeeds << <blocks, threads >> > (d_puzzleImageRGB, d_seeds, d_closestSeed, width, height, Nseeds, ratio);
		ResetSeeds << <blocksReset, threadsReset >> > (d_seeds, Nseeds);
		UpdateSeeds << <blocks, threads >> > (d_puzzleImageRGB, d_closestSeed, d_seeds, width, height, Nseeds);
		AverageSeeds << <blocksReset, threadsReset >> > (d_seeds, Nseeds);
	}
	hipMemcpy(pixelSuperSeed, d_closestSeed, width * height * sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(seeds, d_seeds, sizeof(SeedSuperPixel) * Nseeds, hipMemcpyDeviceToHost);
	hipFree(d_puzzleImageRGB);
	hipFree(d_seeds);
	hipFree(d_closestSeed);
}

void ComputeVoronoiDiagram(SeedSuperPixel* seeds, int* pixelSuperSeed, const int width, const int height, const int Nseeds)
{
	SeedSuperPixel* d_seeds;
	hipMalloc((void**)&d_seeds, sizeof(SeedSuperPixel) * Nseeds);
	hipMemcpy(d_seeds, seeds, sizeof(SeedSuperPixel) * Nseeds, hipMemcpyHostToDevice);

	int* d_closestSeed;
	hipMalloc((void**)&d_closestSeed, width * height * sizeof(int));
	hipMemset(d_closestSeed, 0, width * height * sizeof(int));

	dim3 threads(16, 16);
	dim3 blocks((width + threads.x - 1) / (threads.x), (height + threads.y - 1) / (threads.y));

	ComputeSpatialDistancePixelSeeds<<<blocks,threads>>>(d_seeds, d_closestSeed, width, height, Nseeds);
	hipMemcpy(pixelSuperSeed, d_closestSeed, width * height * sizeof(int), hipMemcpyDeviceToHost);
	hipFree(d_seeds);
	hipFree(d_closestSeed);
}
